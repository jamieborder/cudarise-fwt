
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void FWT(float *fi, float *Fa, int *seq, const int Pa,
        const int Na, const int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // thread Id

    float F1; // storing last value
    float F2; // will be shuffled, all threads have one

    int seqi; // where in memory to put value

    // calculate whether mem pull will be made neg
    // [0:1] -> [0:2] -> [-1:1] -> [1:-1]
    // int negMask = (((tid >> 0) & 1LU) * 2 - 1) * -1;
    int negMask;

    // whether to accept shfl this round
    int srcMask;

    // trying to hide memory pull with ops (not anymore lol)
    seqi = seq[tid];

    if (tid < N) {
        // F1 = fi[tid];
        F1 = fi[(tid / 32) * 32 + seqi];
    }

    int Nm = Na/2;
    for(int pm=0;pm<Pa;pm++) {
        // calculate negMask
        negMask = (((tid >> (Pa-pm-1)) & 1LU) * 2 - 1) * -1;    // 1 or -1

        // calculate src mask
        srcMask = ((tid >> (Pa-pm-1)) & 1LU) ^ 1LU; // 0 or 1
        // if (tid == 2) {
            // printf("tid:%d, pm=%d, srcMask=%d, Nm=%d, negMask=%d\n",
                    // tid, pm, srcMask, Nm, negMask);
        // }

        // apply warp shuffle down
        F2 = srcMask * __shfl_down_sync(0xFFFFFFFF, F1, Nm);

        // flip mask
        srcMask ^= 1LU;
        // if (tid == 2) {
            // printf("tid:%d, pm=%d, srcMask=%d, Nm=%d, negMask=%d\n",
                    // tid, pm, srcMask, Nm, negMask);
        // }

        // if (tid == 7) {
            // printf("F1=%f, F2=%f\n", F1, F2);
        // }

        // apply warp shuffle up
        F2 += srcMask * __shfl_up_sync(0xFFFFFFFF, F1, Nm);

        // if (tid == 2) {
            // printf("F1=%f, F2=%f\n", F1, F2);
        // }

        // add to existing warp value, using negMask
        F1 = F1 * negMask + F2;

        // update shfl width
        // Nm <<= 1;
        Nm >>= 1;
    }

    // write to global memory
    if (tid < N) {
        // Fa[(tid / 32) * 32 + seqi] = F1;
        Fa[tid] = F1;
    }

    return;
}

extern "C"
{
    void run_FWT(const int Pa, const int Na, const int N,
            float *fi, float *Fa, int *seq, const int blockDimX,
            const int gridDimX)
    {
        dim3 blockSize(blockDimX, 1, 1);
        dim3 gridSize( gridDimX, 1, 1);

        FWT<<<gridSize, blockSize>>>(fi, Fa, seq, Pa, Na, N);

        hipError_t err = hipDeviceSynchronize();

        if (err != hipSuccess) {
            printf("%s\n", hipGetErrorString(err));
        }
    }
}
