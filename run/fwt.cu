
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void FWT_SHFL(const float *fi, float *Fa, const int *seq,
        const int Pa, const int Na, const int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // thread Id

    float F1; // storing last value
    float F2; // will be shuffled, all threads have one

    int seqi; // where in memory to put value

    // calculate whether mem pull will be made neg
    // [0:1] -> [0:2] -> [-1:1] -> [1:-1]
    int negMask;

    // whether to accept shfl this round
    int srcMask;

    // every thread load first piece of data
    if (tid < N) {
        F1 = fi[tid];
    }

    // memory pull hidden by next ops
    seqi = seq[tid];

    int Nm = Na/2;
    for(int pm=0;pm<Pa;pm++) {
        // calculate negMask
        negMask = (((tid >> (Pa-pm-1)) & 1LU) * 2 - 1) * -1;    // 1 or -1

        // calculate src mask
        srcMask = ((tid >> (Pa-pm-1)) & 1LU) ^ 1LU; // 0 or 1

        // apply warp shuffle down
        F2 = srcMask * __shfl_down_sync(0xFFFFFFFF, F1, Nm);

        // flip mask
        srcMask ^= 1LU;

        // apply warp shuffle up
        F2 += srcMask * __shfl_up_sync(0xFFFFFFFF, F1, Nm);

        // add to existing warp value, using negMask
        F1 = F1 * negMask + F2;

        // update shfl width
        Nm >>= 1;
    }

    // write to global memory
    if (tid < N) {
        Fa[(tid / 32) * 32 + seqi] = F1;
    }

    return;
}

// global memory version
__global__ void FWT_GM(const float *fi, float *Fa, const int *seq,
        const int Pa, const int Na, const int N, float *F1_global)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // thread Id
    int lid = tid % 32;

    float F1; // storing last value
    float F2; // will be shuffled, all threads have one

    int seqi; // where in memory to put value

    // calculate whether mem pull will be made neg
    // [0:1] -> [0:2] -> [-1:1] -> [1:-1]
    int negMask;

    // whether to accept shfl this round
    int srcMask;

    // memory pull hidden by next ops
    seqi = seq[tid];

    // every thread load first piece of data
    if (tid < N) {
        F1 = fi[tid];
    }

    // now write all data to global memory so all threads can get it
    F1_global[tid] = F1;

    int Nm = Na/2;
    for(int pm=0;pm<Pa;pm++) {
        // calculate negMask
        negMask = (((tid >> (Pa-pm-1)) & 1LU) * 2 - 1) * -1;    // 1 or -1

        // calculate src mask
        srcMask = ((tid >> (Pa-pm-1)) & 1LU) ^ 1LU; // 0 or 1

        // apply warp shuffle down
        F2 = srcMask * F1_global[(tid / 32) * 32 + (lid + Nm) % 32];

        // flip mask
        srcMask ^= 1LU;

        // apply warp shuffle up
        F2 += srcMask * F1_global[(tid / 32) * 32 + (lid - Nm) % 32];

        // add to existing warp value, using negMask
        F1 = F1 * negMask + F2;

        // and write to global memory
        F1_global[tid] = F1;

        // update shfl width
        Nm >>= 1;
    }

    // write to global memory
    if (tid < N) {
        Fa[(tid / 32) * 32 + seqi] = F1;
    }

    return;
}

// shared memory version
__global__ void FWT_SM(const float *fi, float *Fa, const int *seq,
        const int Pa, const int Na, const int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // thread Id
    int lid = tid % 32;

    extern __shared__ float F1_shared[];

    float F1; // storing last value
    float F2; // will be shuffled, all threads have one

    int seqi; // where in memory to put value

    // calculate whether mem pull will be made neg
    // [0:1] -> [0:2] -> [-1:1] -> [1:-1]
    int negMask;

    // whether to accept shfl this round
    int srcMask;

    // memory pull hidden by next ops
    seqi = seq[tid];

    // every thread load first piece of data
    if (tid < N) {
        F1 = fi[tid];
    }

    // now write all data to global memory so all threads can get it
    F1_shared[lid] = F1;

    int Nm = Na/2;
    for(int pm=0;pm<Pa;pm++) {
        // calculate negMask
        negMask = (((tid >> (Pa-pm-1)) & 1LU) * 2 - 1) * -1;    // 1 or -1

        // calculate src mask
        srcMask = ((tid >> (Pa-pm-1)) & 1LU) ^ 1LU; // 0 or 1

        // apply warp shuffle down
        F2 = srcMask * F1_shared[(lid + Nm) % 32];

        // flip mask
        srcMask ^= 1LU;

        // apply warp shuffle up
        F2 += srcMask * F1_shared[(lid - Nm + 32) % 32];

        // add to existing warp value, using negMask
        F1 = F1 * negMask + F2;

        // and write to global memory
        F1_shared[lid] = F1;

        // update shfl width
        Nm >>= 1;
    }

    // write to global memory
    if (tid < N) {
        Fa[(tid / 32) * 32 + seqi] = F1;
    }

    return;
}

extern "C"
{
    void run_FWT_SHFL(const int Pa, const int Na, const int N, const float *fi,
            float *Fa, const int *seq, const int blockDimX, const int gridDimX)
    {
        dim3 blockSize(blockDimX, 1, 1);
        dim3 gridSize( gridDimX, 1, 1);

        FWT_SHFL<<<gridSize, blockSize>>>(fi, Fa, seq, Pa, Na, N);

        hipError_t err = hipDeviceSynchronize();

        if (err != hipSuccess) {
            printf("%s\n", hipGetErrorString(err));
        }
    }

    void run_FWT_GM(const int Pa, const int Na, const int N, const float *fi,
            float *Fa, const int *seq, float *vec, const int blockDimX,
            const int gridDimX)
    {
        dim3 blockSize(blockDimX, 1, 1);
        dim3 gridSize( gridDimX, 1, 1);

        FWT_GM<<<gridSize, blockSize>>>(fi, Fa, seq, Pa, Na, N, vec);

        hipError_t err = hipDeviceSynchronize();

        if (err != hipSuccess) {
            printf("%s\n", hipGetErrorString(err));
        }
    }

    void run_FWT_SM(const int Pa, const int Na, const int N, const float *fi,
            float *Fa, const int *seq, const int blockDimX, const int gridDimX,
            const int sMemSize)
    {
        dim3 blockSize(blockDimX, 1, 1);
        dim3 gridSize( gridDimX, 1, 1);

        FWT_SM<<<gridSize, blockSize, sMemSize>>>(fi, Fa, seq, Pa, Na, N);

        hipError_t err = hipDeviceSynchronize();

        if (err != hipSuccess) {
            printf("%s\n", hipGetErrorString(err));
        }
    }
}
